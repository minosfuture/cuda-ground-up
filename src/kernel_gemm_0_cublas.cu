#include "kernel_gemm.h"

#include <hipblas.h>

#define CUBLAS_CHECK(status)                                                   \
  do {                                                                         \
    hipblasStatus_t error = status;                                             \
    if (error != HIPBLAS_STATUS_SUCCESS) {                                      \
      std::cerr << "cuBLAS error: " << __FILE__ << ":" << __LINE__ << " ";     \
      switch (status) {                                                        \
      case HIPBLAS_STATUS_NOT_INITIALIZED:                                      \
        std::cerr << "HIPBLAS_STATUS_NOT_INITIALIZED\n";                        \
        break;                                                                 \
      case HIPBLAS_STATUS_ALLOC_FAILED:                                         \
        std::cerr << "HIPBLAS_STATUS_ALLOC_FAILED\n";                           \
        break;                                                                 \
      case HIPBLAS_STATUS_INVALID_VALUE:                                        \
        std::cerr << "HIPBLAS_STATUS_INVALID_VALUE\n";                          \
        break;                                                                 \
      case HIPBLAS_STATUS_ARCH_MISMATCH:                                        \
        std::cerr << "HIPBLAS_STATUS_ARCH_MISMATCH\n";                          \
        break;                                                                 \
      case HIPBLAS_STATUS_EXECUTION_FAILED:                                     \
        std::cerr << "HIPBLAS_STATUS_EXECUTION_FAILED\n";                       \
        break;                                                                 \
      case HIPBLAS_STATUS_INTERNAL_ERROR:                                       \
        std::cerr << "HIPBLAS_STATUS_INTERNAL_ERROR\n";                         \
        break;                                                                 \
      default:                                                                 \
        std::cerr << "Unknown error\n";                                        \
      }                                                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

void kernel_gemm_0_launch(GemmData &data, const unsigned int num_runs) {
  hipblasHandle_t handle;
  CUBLAS_CHECK(hipblasCreate(&handle));

  // warmup
  auto kernel_func = [&]() {
    CUBLAS_CHECK(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, data.dim_n,
                             data.dim_m, data.dim_k, &data.alpha, data.dev_B,
                             data.dim_n, data.dev_A, data.dim_k, &data.beta,
                             data.dev_C, data.dim_n));
  };
  kernel_func();
  data.set_c_ref();

  KernelProfiler profiler;
  for (int i = 0; i < num_runs; i++) {
    profiler.start();
    kernel_func();
    profiler.stop();
  }
  CUDA_CHECK(hipPeekAtLastError());

  std::cout << "kernel 0 (cublas) GFLOPS for size (" << data.dim_m << "x"
            << data.dim_n << "x" << data.dim_k << "): "
            << profiler.log_gemm_stats(data.dim_m, data.dim_n, data.dim_k)
            << std::endl;
}
