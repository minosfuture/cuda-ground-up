#include "hip/hip_runtime.h"
#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

constexpr int kBlockSize = 512;
constexpr int kTileCount = 32; // number of elements processed by each thread
constexpr int kPerBlockDataSize = kTileCount * kBlockSize;

__global__ void kernel_vector_add_2(half *A, half *B, half *C,
                                    const unsigned int N) {
  const int start_idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = 0; i < kTileCount; i++) {
    int idx = start_idx + i * kBlockSize;
    C[idx] = A[idx] + B[idx];
  }
}

void kernel_vector_add_2_launch(const unsigned int N,
                                const unsigned int num_runs = 10) {
  half *dev_A, *dev_B, *dev_C;
  CUDA_CHECK(hipMalloc(&dev_A, N * sizeof(half)));
  CUDA_CHECK(hipMalloc(&dev_B, N * sizeof(half)));
  CUDA_CHECK(hipMalloc(&dev_C, N * sizeof(half)));

  dim3 grid_dim(N / kPerBlockDataSize);
  dim3 block_dim(kBlockSize);

  KernelProfiler profiler;
  for (int i = 0; i < num_runs; i++) {
    profiler.start();
    kernel_vector_add_2<<<grid_dim, block_dim, 0, hipStreamPerThread>>>(
        dev_A, dev_B, dev_C, N);
    profiler.stop();
  }
  CUDA_CHECK(hipPeekAtLastError());

  std::cout << __FUNCTION__ << " GFLOPS for size (" << N
            << "): " << profiler.logVectorAddKernelStats(N) << std::endl;
}
